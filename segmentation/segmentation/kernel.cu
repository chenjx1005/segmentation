#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "gpu_common.h"

#define BLOCK_SIZE 16

__device__ float CalDistance(const unsigned char a[3], const unsigned char b[3])
{
	return sqrt(pow(float(a[0] - b[0]),2) + pow(float(a[1] - b[1]),2) + pow(float(a[2] - b[2]),2));
}

__global__ void MatCopy(int *dst, const unsigned char *src, int step, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	size_t s = i * step + j;
	if (s < size)
		dst[s] = src[s];
}

__global__ void VecAdd(float **A, float **B, float **C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < 1000 && j < 1000)
		C[i][j] = A[i][j] + B[i][j];
}

__global__ void DifferenceKernel(const unsigned char (*color)[3], 
									  const unsigned char *depth, 
									  double (*diff)[8],
									  size_t rows, 
									  size_t cols,
									  int (*record)[8]);

__global__ void SumKernel(const double *diff,
						  double *odiff,
						  const int *record,
						  unsigned int *orecord,
						  size_t n);

int mymain()
{
	clock_t t;
	int i,j;
	unsigned char a[1000];
	for(i = 0; i < 1000; i++) a[i]=i%256;
	int b[1000];
	int *d_C;
    hipMalloc(&d_C, 1000*sizeof(int));
	unsigned char *d_A;
	hipMalloc(&d_A, 1000);
	hipMemcpy(d_A, a, 1000, hipMemcpyHostToDevice);
	dim3 threadsPerBlock(16,16);
	dim3 numBlocks((1000+16-1)/threadsPerBlock.x, (1000+16-1)/threadsPerBlock.y);
	t = clock();
	MatCopy<<<numBlocks, threadsPerBlock>>>(d_C, d_A, 10, 1000);
	t = clock() - t;
	hipMemcpy(b, d_C, 1000*sizeof(int), hipMemcpyDeviceToHost);
	for(i = 0; i < 1000; i++) printf("%d ",b[i]);
	printf("run time is %f seconds\n", ((float)t)/CLOCKS_PER_SEC);
	return 0;
}



hipError_t ComputeDifferenceWithCuda(const unsigned char (*color)[3], 
									  const unsigned char *depth, 
									  double (*diff)[8],
									  size_t rows, 
									  size_t cols)
{
	//load color and depth to device memory
	unsigned char (*d_color)[3];
	size_t size = rows*cols*3;
	hipMalloc(&d_color, size);
	hipMemcpy(d_color, color, size, hipMemcpyHostToDevice);
	unsigned char *d_depth;
	size = rows*cols;
	hipMalloc(&d_depth, size);
	hipMemcpy(d_depth, depth, size, hipMemcpyHostToDevice);

	//Allocate diff in device memory
	double (*d_diff)[8];
	size = rows * cols * 8 * sizeof(double);
	hipMalloc(&d_diff, size);

	//Allocate depth > 30 pixels record in device memory
	int (*d_record)[8];
	size = rows * cols * 8 * sizeof(int);
	hipMalloc(&d_record, size);

	//Invoke kernel
	dim3 dimBlock(BLOCK_SIZE/2, BLOCK_SIZE/2, 8);
	dim3 dimGrid((cols + BLOCK_SIZE - 1)/dimBlock.x, (rows + BLOCK_SIZE - 1)/dimBlock.y);
	DifferenceKernel<<<dimGrid, dimBlock>>>(d_color, d_depth, d_diff, rows, cols, d_record);

	int block_sum_size = BLOCK_SIZE * BLOCK_SIZE;
	size_t n = rows * cols * 8;
	int block_sum_num = ((n + block_sum_size - 1) / block_sum_size + 1) / 2;
	double *odiff;
	unsigned int *orecord;
	size_t sum_size = block_sum_num * sizeof(double);
	size_t count_size = block_sum_num * sizeof(unsigned int);
	hipMalloc(&odiff, sum_size);
	hipMalloc(&orecord, count_size);
	SumKernel<<<block_sum_num, block_sum_size>>>((const double *)d_diff, odiff, (const int *)d_record, orecord, n);
	
	//Read diff from device memory
	size = rows * cols * 8 * sizeof(double);
	hipMemcpy(diff, d_diff, size, hipMemcpyDeviceToHost);
	
	//Read sum and count from device memory
	double *cpu_sum = (double *)malloc(sum_size);
	unsigned int *cpu_count = (unsigned int *)malloc(count_size);
	hipMemcpy(cpu_sum, odiff, sum_size, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_count, orecord, count_size, hipMemcpyDeviceToHost);

	//Free device memory
	hipFree(d_color);
	hipFree(d_depth);
	hipFree(d_diff);
	hipFree(d_record);
	hipFree(odiff);
	hipFree(orecord);
	//Free host memory
	free(cpu_sum);
	free(cpu_count);

	return hipSetDevice(0);
}

__global__ void DifferenceKernel(const unsigned char (*color)[3], 
					  const unsigned char *depth, 
					  double (*diff)[8],
					  size_t rows, 
					  size_t cols,
					  int (*record)[8])
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int k = threadIdx.z;
	int limit_i, limit_j;
	int current = i * cols + j;
	int next;
	if (i < rows && j < cols)
	{
		switch (k)
		{
		case 0:
			limit_i = -1;
			limit_j = 0;
			next = i*cols+j-1;
			break;
		case 1:
			limit_i = -1;
			limit_j = cols-1;
			next = i*cols+j+1;
			break;
		case 2:
			limit_i = 0;
			limit_j = -1;
			next = (i-1)*cols+j;
			break;
		case 3:
			limit_i = rows-1;
			limit_j = -1;
			next = (i+1)*cols+j;
			break;
		case 4:
			limit_i = 0;
			limit_j = 0;
			next = (i-1)*cols+j-1;
			break;
		case 5:
			limit_i = rows-1;
			limit_j = cols-1;
			next = (i+1)*cols+j+1;
			break;
		case 6:
			limit_i = 0;
			limit_j = cols-1;
			next = (i-1)*cols+j+1;
			break;
		case 7:
			limit_i = rows-1;
			limit_j = 0;
			next = (i+1)*cols+j-1;
			break;
		}
		if(i==limit_i || j==limit_j)
		{
			record[current][k] = 0;
			diff[current][k] = 0;
		}
		else
		{
			diff[current][k] = CalDistance(color[current], color[next]);
			if (abs(float(depth[current] - depth[next]))>30)
				record[current][k] = -1;
			else
				record[current][k] = 1;
		}
	}
}

__global__ void SumKernel(const double *diff,
						  double *odiff,
						  const int *record,
						  unsigned int *orecord, 
						  size_t n)
{
	extern __shared__ double diffsum[];
	extern __shared__ int count[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

	double mysum, mycount;
	if(i < n) 
	{
		mysum = diff[i];
		mycount = abs(record[i]);
		if(i + blockDim.x < n)
		{
			mysum += diff[i + blockDim.x];
			mycount += abs(record[i + blockDim.x]);
		}
	}
	else mysum = mycount = 0;
	
	diffsum[tid] = mysum;
	count[tid] = mycount;
	__syncthreads();

	for(unsigned int s = blockDim.x/2; s > 0; s>>1)
	{
		if(tid < s)
		{
			diffsum[tid] += diffsum[tid + s];
			count[tid] += abs(count[tid + s]);
		}
		__syncthreads();
	}
	if(tid == 0) odiff[blockIdx.x] = diffsum[0], orecord[blockIdx.x] = count[0];
}