#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <float.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "gpu_common.h"

#define BLOCK_SIZE 16
#define MAX_J 250.0
#define SPIN 256
#define kK = 1.3806488e-4

static unsigned char (*d_color)[3] = 0;
static unsigned char *d_depth = 0;
static float (*d_diff)[8] = 0;
static int (*d_record)[8] = 0;
static float *odiff = 0;
static unsigned int *orecord = 0;
static float *cpu_sum = 0;
static unsigned int *cpu_count = 0;
static unsigned char *d_states = 0;
static hiprandState *devStates = 0;
static unsigned int *rand_value = 0;
static unsigned char *d_boundry = 0;
//Optical Flow
static unsigned char *new_depth = 0;
static unsigned char *new_states = 0;

void time_print(char *info, int flag)
{
	static clock_t t = clock();
	if(flag)
		printf("%s run time is %f ms\n", info, (clock() - (float)t)/CLOCKS_PER_SEC * 1000);
	t = clock();
}

__device__ float CalDistance(const unsigned char a[3], const unsigned char b[3])
{
	return sqrt(pow(float(a[0] - b[0]),2) + pow(float(a[1] - b[1]),2) + pow(float(a[2] - b[2]),2));
}

__global__ void setup_kernel(hiprandState *state) 
{ 
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(1234, id, 0, &state[id]);
}

__global__ void generate(unsigned int *rand_value, hiprandState *state, int rows)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState s = state[id];
	for(int i = 0; i < rows * 3; i++)
	  rand_value[id * rows * 3 + i]	= hiprand(&s);
	state[id] = s;
}

__global__ void DifferenceKernel(const unsigned char (*color)[3], 
									  const unsigned char *depth, 
									  float (*diff)[8],
									  size_t rows, 
									  size_t cols,
									  int (*record)[8]);

__global__ void SumKernel(const float *diff,
						  float *odiff,
						  const int *record,
						  unsigned int *orecord,
						  size_t n);

__global__ void DecorateDiff(float *diff, const int *record, float mean, float max_j);

__global__ void Metropolis(const float (*diff)[8], unsigned char *states, int x, int y, int rows, int cols, float t, unsigned int *rand_value);

__global__ void BoundryKernel(const unsigned char *states, unsigned char *boundry, int rows, int cols);

__global__ void LoadNextKernel(unsigned char *states, const unsigned char *old_states, const unsigned char *depth, const unsigned char *old_depth, cv::gpu::PtrStep<float> flow_x, cv::gpu::PtrStep<float> flow_y, int rows, int cols);

__global__ void LoadNextUpdateKernel(unsigned char *states, int rows, int cols);

hipError_t CudaSetup(size_t rows, size_t cols)
{
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//Allocate GPU color and depth to device memory
	size_t size = rows*cols*3;
	cudaStatus = hipMalloc(&d_color, size);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc color failed!");
		goto Error;	
	}

	size = rows*cols;
	cudaStatus = hipMalloc(&d_depth, size);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc depth failed!");
		goto Error;	
	}

	//Allocate GPU diff
	size = rows * cols * 8 * sizeof(float);
	cudaStatus = hipMalloc(&d_diff, size);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc diff failed!");
		goto Error;	
	}

	//Allocate GPU record when depth > 30
	size = rows * cols * 8 * sizeof(int);
	cudaStatus = hipMalloc(&d_record, size);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc record failed!");
		goto Error;	
	}

	//Allocate GPU spin states and boundry
	size = rows * cols;
	cudaStatus = hipMalloc(&d_states, size);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc spin states failed!");
		goto Error;	
	}

	cudaStatus = hipMalloc(&d_boundry, size);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc boundry failed!");
		goto Error;	
	}

	cudaStatus = hipMalloc(&devStates, cols * sizeof(hiprandState));
	cudaStatus = hipMemset(devStates, 0, cols * sizeof(hiprandState));

	setup_kernel<<<cols / 64, 64>>>(devStates);

	cudaStatus = hipMalloc(&rand_value, rows * cols * 3 * sizeof(unsigned int));

	if (cudaStatus != hipSuccess) {
		printf("hipMalloc random value failed!");
		goto Error;	
	}

	//Allocate GPU sum
	size_t block_sum_size = BLOCK_SIZE * BLOCK_SIZE;
	size_t num = ((rows * cols * 8 + block_sum_size - 1) / block_sum_size + 1) / 2;
	size = num * sizeof(float);
	cudaStatus = hipMalloc(&odiff, size);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc Sumdiff failed!");
		goto Error;	
	}
	cpu_sum = (float *)malloc(size);
	if (cpu_sum == NULL) {
		printf("Malloc cpu_sum failed!");
		goto Error;	
	}

	size = num * sizeof(unsigned int);
	cudaStatus = hipMalloc(&orecord, size);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc Sumrecord failed!");
		goto Error;	
	}
	cpu_count = (unsigned int *)malloc(size);
	if (cpu_count == NULL) {
		printf("Malloc cpu_count failed!");
		goto Error;	
	}

	//Allocate GPU Optical Flow
	size = rows * cols;
	cudaStatus = hipMalloc(&new_depth, size);
	cudaStatus = hipMalloc(&new_states, size);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc Optical Flow failed!");
		goto Error;	
	}

	return cudaStatus;

Error:
	CudaRelease();
	return cudaStatus;
}

void CudaRelease()
{
	if (d_color) hipFree(d_color);
	if (d_depth) hipFree(d_depth);
	if (d_diff) hipFree(d_diff);
	if (d_record) hipFree(d_record);
	if (odiff) hipFree(odiff);
	if (orecord) hipFree(orecord);
	if (cpu_count) hipFree(cpu_count);
	if (d_states) hipFree(d_states);
	if (devStates) hipFree(devStates);
	if (rand_value) hipFree(rand_value);
	if (d_boundry) hipFree(d_boundry);
	if (cpu_sum) free(cpu_sum);
	if (new_depth) hipFree(new_depth);
	if (new_states) hipFree(new_states);

	d_color = NULL;
	d_depth = NULL;
	d_diff = NULL;
	d_record = NULL;
	odiff = NULL;
	orecord = NULL;
	cpu_count = NULL;
	d_states = NULL;
	devStates = NULL;
	rand_value = NULL;
	d_boundry = NULL;
	cpu_sum = 0;
	new_depth = NULL;
	new_states = NULL;
}

void ComputeDifferenceWithCuda(const unsigned char (*color)[3], 
									  const unsigned char *depth, 
									  float (*diff)[8],
									  size_t rows, 
									  size_t cols)
{
	time_print("", 0);

	hipMemcpy(d_color, color, rows * cols * 3, hipMemcpyHostToDevice);
	//Copy depth for the first frame
	//Other frames copy depth in LoadNextFrameWithCuda function 
	static int re = 0;

	if(!re++)
		hipMemcpy(d_depth, depth, rows * cols, hipMemcpyHostToDevice);
	time_print("GPU Copy");

	//Invoke kernel
	dim3 dimBlock(BLOCK_SIZE/2, BLOCK_SIZE/2, 8);
	dim3 dimGrid((cols + BLOCK_SIZE - 1)/dimBlock.x, (rows + BLOCK_SIZE - 1)/dimBlock.y);
	DifferenceKernel<<<dimGrid, dimBlock>>>(d_color, d_depth, d_diff, rows, cols, d_record);
	
	time_print("GPU difference kernel");

	//Compute Block Count and Sum
	int block_sum_size = BLOCK_SIZE * BLOCK_SIZE;
	size_t n = rows * cols * 8;
	int block_sum_num = ((n + block_sum_size - 1) / block_sum_size + 1) / 2;
	size_t sum_size = block_sum_num * sizeof(float);
	size_t count_size = block_sum_num * sizeof(unsigned int);
	
	SumKernel<<<block_sum_num, block_sum_size>>>((const float *)d_diff, odiff, (const int *)d_record, orecord, n);

	//Read sum and count from device memory
	hipMemcpy(cpu_count, orecord, count_size, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_sum, odiff, sum_size, hipMemcpyDeviceToHost);

	//compute sum and mean
	int i, count = 0;
	float sum = 0;
	for(i = 0; i < block_sum_num; i++) sum += cpu_sum[i], count += cpu_count[i];
	float mean = sum / count;
	printf("sum is %lf, count is %d, mean_diff is %lf when alpha=%lf\n", sum, count, mean, 1.0);
	time_print("GPU Compute Sum and Mean");

	//Decorate diff
	if (mean < FLT_EPSILON) mean = 1.0;
	int dec_num = (n + block_sum_size - 1) / block_sum_size;
	DecorateDiff<<<dec_num, block_sum_size>>>((float *)d_diff, (const int *)d_record, mean, MAX_J);

	//Read diff from device memory
	hipMemcpy(diff, d_diff, rows * cols * 8 * sizeof(float), hipMemcpyDeviceToHost);

	time_print("GPU DECORATE");
}

void MetropolisOnceWithCuda(float t, unsigned char *states, int rows, int cols)
{
	static int n = 0;

	if(!n++)
		hipMemcpy(d_states, states, rows * cols, hipMemcpyHostToDevice);

	generate<<<cols / 64, 64>>>((unsigned int *)rand_value, devStates, rows);

	dim3 block_num(8,8,8);
	dim3 grid_num(((cols+1)/2+7)/8, ((rows+1)/2+7)/8, 1); 
	Metropolis<<<grid_num, block_num>>>(d_diff, d_states, 0, 0, rows, cols, t, rand_value); 
	Metropolis<<<grid_num, block_num>>>(d_diff, d_states, 0, 1, rows, cols, t, rand_value);
	Metropolis<<<grid_num, block_num>>>(d_diff, d_states, 1, 0, rows, cols, t, rand_value);
	Metropolis<<<grid_num, block_num>>>(d_diff, d_states, 1, 1, rows, cols, t, rand_value);

	hipMemcpy(states, d_states, rows*cols, hipMemcpyDeviceToHost);
}

void GenBoundryWithCuda(unsigned char *boundry, int rows, int cols)
{
	hipMemset(d_boundry, 255, rows * cols);
	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_size((cols+BLOCK_SIZE-1) / BLOCK_SIZE, (rows+BLOCK_SIZE-1) / BLOCK_SIZE);

	BoundryKernel<<<grid_size, block_size>>>(d_states, d_boundry, rows, cols);
	hipMemcpy(boundry, d_boundry, rows*cols, hipMemcpyDeviceToHost);
}

void CopyStatesToDevice(unsigned char *states, int rows, int cols)
{
	hipMemcpy(d_states, states, rows * cols, hipMemcpyHostToDevice);
}

void LoadNextFrameWithCuda(unsigned char *states, const unsigned char *depth, cv::gpu::PtrStep<float> flow_x, cv::gpu::PtrStep<float> flow_y, int rows, int cols)
{
	size_t size = rows * cols;
	hipMemset(new_states, 0, size);
	hipMemcpy(new_depth, depth, size, hipMemcpyHostToDevice);	

	//Kernel
	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_size((cols+BLOCK_SIZE-1) / BLOCK_SIZE, (rows+BLOCK_SIZE-1) / BLOCK_SIZE);

	LoadNextKernel<<<grid_size, block_size>>>(new_states, d_states, new_depth, d_depth, flow_x, flow_y, rows, cols);
	LoadNextUpdateKernel<<<grid_size, block_size>>>(new_states, rows, cols);

	//swap new and old memory
	unsigned char *tmp;
	tmp = d_states;
	d_states = new_states;
	new_states = tmp;

	tmp = d_depth;
	d_depth = new_depth;
	new_depth = tmp;

	//Copy states to Host
	hipMemcpy(states, d_states, size, hipMemcpyDeviceToHost);
}

__global__ void DifferenceKernel(const unsigned char (*color)[3], 
					  const unsigned char *depth, 
					  float (*diff)[8],
					  size_t rows, 
					  size_t cols,
					  int (*record)[8])
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int k = threadIdx.z;
	int limit_i, limit_j;
	int current = i * cols + j;
	int next;
	float val=0;
	if (i < rows && j < cols)
	{
		switch (k)
		{
		case 0:
			limit_i = -1;
			limit_j = 0;
			next = i*cols+j-1;
			break;
		case 1:
			limit_i = -1;
			limit_j = cols-1;
			next = i*cols+j+1;
			break;
		case 2:
			limit_i = 0;
			limit_j = -1;
			next = (i-1)*cols+j;
			break;
		case 3:
			limit_i = rows-1;
			limit_j = -1;
			next = (i+1)*cols+j;
			break;
		case 4:
			limit_i = 0;
			limit_j = 0;
			next = (i-1)*cols+j-1;
			break;
		case 5:
			limit_i = rows-1;
			limit_j = cols-1;
			next = (i+1)*cols+j+1;
			break;
		case 6:
			limit_i = 0;
			limit_j = cols-1;
			next = (i-1)*cols+j+1;
			break;
		case 7:
			limit_i = rows-1;
			limit_j = 0;
			next = (i+1)*cols+j-1;
			break;
		}
		if(i==limit_i || j==limit_j)
		{
			record[current][k] = 0;
			diff[current][k] = 0;
		}
		else
		{
			val = CalDistance(color[current], color[next]);
			diff[current][k] = val;
			if (abs(float(depth[current] - depth[next]))>30)
				record[current][k] = -1;
			else
				record[current][k] = 1;
		}
	}
}

__global__ void SumKernel(const float *diff,
						  float *odiff,
						  const int *record,
						  unsigned int *orecord, 
						  size_t n)
{
	__shared__ float diffsum[BLOCK_SIZE * BLOCK_SIZE];
	__shared__ int count[BLOCK_SIZE * BLOCK_SIZE];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

	float mysum;
	int mycount;
	if(i < n) 
	{
		mysum = diff[i];
		mycount = abs(record[i]);
		if(i + blockDim.x < n)
		{
			mysum += diff[i + blockDim.x];
			mycount += abs(record[i + blockDim.x]);
		}
	}
	else mysum = mycount = 0;
	
	count[tid] = mycount;
	diffsum[tid] = mysum;
	__syncthreads();

	for(unsigned int s = blockDim.x/2; s > 0; s>>=1)
	{
		if(tid < s)
		{
			diffsum[tid] += diffsum[tid + s];
			count[tid] += abs(count[tid + s]);
		}
		__syncthreads();
	}
	if(tid == 0)
	{
		odiff[blockIdx.x] = diffsum[0];
		orecord[blockIdx.x] = count[0];
	}
}

__global__ void DecorateDiff(float *diff, const int *record, float mean, float max_j)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(record[i] == 1)
		diff[i] = diff[i] * (1.0 / mean) - 1.0;
	else if(record[i] == -1)
		diff[i] = max_j;
}

__global__ void Metropolis(const float (*diff)[8], unsigned char *states, int x, int y, int rows, int cols, float t, unsigned int *rand_value)
{
	__shared__ float energy[8][8][8]; 

	const int P[8][2] = {{0,-1}, {0,1}, {-1,0}, {1,0}, {-1,-1}, {1,1}, {-1,1}, {1,-1}};
	int i;
	//TODO: out of rows and cols range
	int p_i = blockIdx.y * blockDim.y + threadIdx.y;
	int p_j = blockIdx.x * blockDim.x + threadIdx.x;
	int z = threadIdx.z;

	int b_i = threadIdx.y;
	int b_j = threadIdx.x;

	//Compute real position
	p_i = x + 2 * p_i;
	p_j = y + 2 * p_j;

	//Read global memory position into thread
	const float *d = diff[p_i*cols + p_j];
	
	int ki, kj;
	ki = p_i+P[z][0];
	kj = p_j+P[z][1];
	if (ki < 0 || ki >= rows || kj < 0 || kj >= cols)
	{
		energy[b_i][b_j][z] = FLT_MAX;
	}
	else
	{
		unsigned char s = states[ki * cols + kj];
		float e = 0;
		for (i = 0; i < 8; i++)
			{
				ki = p_i + P[i][0];
				kj = p_j + P[i][1];
				if (ki < 0 || ki >= rows || kj < 0 || kj >= cols) continue;
				e += d[i] * (s == states[ki*cols+kj]);
			}
		energy[b_i][b_j][z] = e;
	}
	__syncthreads();

	//find min energy
	if(z == 0)
	{
		unsigned int id = (p_i * cols + p_j) * 3;
		unsigned char current_s = states[p_i * cols + p_j];
		float current_e = 0;
		float min_e = 0;
		unsigned char min_s, local_s;
		unsigned int r = rand_value[id];
		for (i = 0; i < 8; i++)
		{
			//compute current energy
			ki = p_i + P[i][0];
			kj = p_j + P[i][1];
			if (ki < 0 || ki >= rows || kj < 0 || kj >= cols) continue;
			local_s = states[ki*cols + kj];
			current_e += d[i] * (current_s == local_s);
			//if set current state a neibor state, compute the new energy
			if(local_s == current_s) continue;
			int compare = 2;
			if (energy[b_i][b_j][i] < min_e)
			{
				min_e = energy[b_i][b_j][i];
				min_s = local_s;	
				compare = 2;
			}
			else if (energy[b_i][b_j][i] == min_e)
			{
				if (r % 100 / 100.0 < 1.0 / compare)
				{
					min_s = local_s;
					compare++;
				}
				r /= 10;
			}
		}
		float diff_e = min_e - current_e;
		float r1 =  (rand_value[id + 1] % 1000) / 1000.0;
		float r2 = exp(-1 * diff_e / (t * 1.38064e-4));
		if (diff_e <= 0 || r1 < r2)
		{
			if (fabs(min_e) < FLT_EPSILON)
			{
				//TODO: Not use exist spin number around the pxiel
				min_s = (unsigned char)rand_value[id + 2];
			}
				states[p_i*cols + p_j] = min_s;
		}
	}
}

__global__ void BoundryKernel(const unsigned char *states, unsigned char *boundry, int rows, int cols)
{
	__shared__ unsigned char shared_states[BLOCK_SIZE][BLOCK_SIZE];

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int t_i = threadIdx.y;
	int t_j = threadIdx.x;

	unsigned char s = states[i * cols + j];

	if (i < rows && j < cols)
	{
		shared_states[t_i][t_j] = s;
		__syncthreads();

		unsigned char pre0;
		unsigned char pre1;
		unsigned char b = 255;

		if (t_j > 1) pre0 = shared_states[t_i][t_j -2], pre1 = shared_states[t_i][t_j - 1];
		else if (t_j == 1) pre0 = (j == 1 ? s : states[i * cols + j - 2]), pre1 = shared_states[t_i][t_j - 1];
		else pre0 = (j == 0 ? s : states[i * cols + j - 2]), pre1 = (j == 0 ? s : states[i * cols + j - 1]);

		if (pre0 != pre1 && pre1 != s) b = 0;
		else
		{
			if (t_i > 1) pre0 = shared_states[t_i - 2][t_j], pre1 = shared_states[t_i - 1][t_j];
			else if (t_i == 1) pre0 = (i == 1 ? s : states[(i-2) * cols + j]), pre1 = shared_states[t_i - 1][t_j];
			else pre0 = (i == 0 ? s : states[(i-2) * cols + j]), pre1 = (i == 0 ? s : states[(i-1) * cols + j]);
			if (pre0 != pre1 && pre1 != s) b = 0;
		}

		boundry[i * cols + j] = b;
	}
}

__global__ void LoadNextKernel(unsigned char *states, const unsigned char *old_states, const unsigned char *depth, const unsigned char *old_depth, cv::gpu::PtrStepf flow_x, cv::gpu::PtrStepf flow_y, int rows, int cols)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < rows && j < cols)
	{
		int x = i + (int)flow_x.ptr(i)[j];
		int y = j + (int)flow_y.ptr(i)[j];
		if (x >= 0 && y >=0 && x < rows && y < cols &&
			abs((int)depth[x * cols + y] - (int)old_depth[i * cols + j]) <= 30)
		{
			states[x * cols + y] = old_states[i * cols + j];
		}
	}
}

__global__ void LoadNextUpdateKernel(unsigned char *states, int rows, int cols)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < rows && j < cols)
	{
		if (states[i * cols + j] == 0)
		{
			states[i * cols + j] = (unsigned char)(i * cols + j);
		}
	}
}